#include "kernels.h"

#include <hip/hip_runtime.h>
#include <thrust/transform.h>
#include <thrust/count.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/execution_policy.h>
#include <thrust/sort.h>

#include <algorithm>
#include <iostream>
#include <cstdlib>
#include <ctime>


int main(int argc, char *argv[])
{
    int default_alg = MIS_ALG;

    if (argc == 2) {
        std::string a = std::string(argv[1]);
        if (a.compare(6, 3, "mis") == 0)
            default_alg = MIS_ALG;
        else if (a.compare(6, 3, "jpl") == 0)
            default_alg = JPL_ALG;
        else if (a.compare(6, 3, "ldf") == 0)
            default_alg = LDF_ALG;
    }
    unsigned int n_nodes, n_values;
    /*unsigned n_edges;*/
    std::cin >> n_nodes >> n_values;
    /*n_values = 2 * n_edges;*/

    /*thrust::host_vector<unsigned int> t_Ax(n_values), t_Ay(n_values), t_Ao(n_nodes+1);*/
    /*thrust::fill(t_Ao.begin(), t_Ao.end(), 0);*/
    /*unsigned int x_cord, y_cord;*/
    /*for (int ti=0; ti<n_values; ti++) {*/
        /*std::cin >> x_cord >> y_cord; */
        /*t_Ax[2*ti] = x_cord;*/
        /*t_Ay[2*ti] = y_cord;*/
        /*t_Ax[2*ti+1] = y_cord;*/
        /*t_Ay[2*ti+1] = x_cord;*/
        /*t_Ao[x_cord+1]++;*/
        /*t_Ao[y_cord+1]++;*/
    /*}*/

    /*thrust::inclusive_scan(t_Ao.begin(), t_Ao.end(), t_Ao.begin());*/

    /*for (int ind = 0; ind < n_nodes+1; ind++) {*/
        /*std::cout << t_Ao[ind] << " " ;*/
    /*}*/
    /*std::cout << "AO" << std::endl;*/

    thrust::host_vector<unsigned int> h_Ao(n_nodes+1), h_Ac(n_values);
    for (int i=0; i<n_nodes+1; i++) {
        std::cin >> h_Ao[i] ;
    }
    for (int i=0; i<n_values; i++) {
        std::cin >> h_Ac[i] ;
    }

    /*thrust::sort_by_key(t_Ax.begin(), t_Ax.end(), t_Ay.begin());*/
    thrust::device_vector<unsigned int> d_Ao = h_Ao, d_Ac = h_Ac;
    thrust::device_vector<int> d_colors(n_nodes);

    /*thrust::fill(d_colors.begin(), d_colors.end(), -1); */

    unsigned int *r_Ao = thrust::raw_pointer_cast(d_Ao.data());
    unsigned int *r_Ac = thrust::raw_pointer_cast(d_Ac.data());
    int *r_c = thrust::raw_pointer_cast(d_colors.data());

    bool *result;
    hipHostAlloc(&result, sizeof(bool), 0);
    *result = true;

    switch(default_alg) {
        
        case MIS_ALG:
            std::cout << "Running MIS_ALG on " << n_nodes << " nodes" << std::endl;
            mis_coloring(n_nodes, r_Ao, r_Ac, r_c);
            break;
        case JPL_ALG:
            std::cout << "Running JPL_ALG on " << n_nodes << " nodes" << std::endl;
            jpl_coloring(n_nodes, r_Ao, r_Ac, r_c);
            break;
        case LDF_ALG:
            std::cout << "Running LDF_ALG on " << n_nodes << " nodes" << std::endl;
            ldf_coloring(n_nodes, r_Ao, r_Ac, r_c);
            break;
        default:
            std::cerr << "Invalid alg" << std::endl;
            return 0;
    }
    /*hipDeviceSynchronize();*/
    check_correctness(n_nodes, r_Ao, r_Ac, r_c, result);

    hipDeviceSynchronize();
    if(*result)
      std::cout << "Check successful " << std::endl;
    else
      std::cout << "Check failed " << std::endl;

    /*for(int i=0; i<n_nodes; i++)*/
        /*std::cout << d_colors[i] << " ";*/
    /*std::cout << std::endl;*/

}
